//
// Created by 40169 on 2022/3/14.
//

#include "include/error.cuh"

__host__ void eventF()
{
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time,start,stop));
    printf("Time = %g ms. \n",elapsed_time);
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}
